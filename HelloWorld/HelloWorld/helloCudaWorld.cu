// Hello Cuda World Program //
/*
* Author: Malhar Bhatt
* Subject : High Performance Computing
*
*/


#include <hip/hip_runtime.h>
#include <iostream>

/**
* Empty Function named Kernel() qualified with __global__
*
*/

__global__ void kernel (void)
{
}

int main(void)
{
	kernel<<<1,1>>>(); // Calling Empty Function
	printf("Hello Cuda World !!!\n"); // Printing Hello Cuda World
	system("pause");
	return 0;

}